#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

#define TILE_WIDTH 100 // for shared kernel
#define DEBUG 0
typedef double Type;

void transpose_CPU(Type* in_, Type* out, int dim1, int dim2){
		
		for(int n = 0; n<dim1*dim2; n++) {
        int i = n/dim1;
        int j = n%dim1;
        out[n] = in_[dim2*j + i];
    }
}

//transpose per element
__global__ void transpose_GPU(Type* in_, Type* out, int dim1, int dim2){

	int tile_size = blockDim.x ;
	int column = tile_size * blockIdx.x + threadIdx.x;
	int row = tile_size * blockIdx.y + threadIdx.y;

	if(column < dim2 && row < dim1){
		out[column*dim2 + row] = in_[column + row*dim2];
	}
}

// SHARED MEM APROACH - use shared memory
__global__ void transpose_GPU_shared(Type* in_, Type* out, int dim1, int dim2){

	// fill data into shared memory
	__shared__ Type M_Shared[TILE_WIDTH][TILE_WIDTH];

	int tile_size =TILE_WIDTH;
	int column = tile_size * blockIdx.x + threadIdx.x;
	int row = tile_size * blockIdx.y + threadIdx.y;
	int index_in = row*dim2 + column;
	int index_out = column*dim2 + row;


	if(row < dim1 && column < dim2 && index_in < dim1*dim2){
		M_Shared[threadIdx.y][threadIdx.x] = in_[index_in];
	}
	__syncthreads(); // wait all other threads to go further.

	if(row < dim1 && column < dim2 && index_out < dim1*dim2){
		out[index_out] = M_Shared[threadIdx.y][threadIdx.x];
	}
}

void print_matrix(Type * mat, int dim1, int dim2){
	for (int i = 0; i < dim1; i++) {
		for (int j = 0; j < dim2; j++) {
			Type num = mat[i + dim2 * j];
			printf(" %.2f ",num);
		}
		printf("\n");
	}
}
void initial_matirx(Type * mat, int dim1, int dim2){
		for (int i = 0; i < dim1*dim2; ++i) {
			mat[i] = i;
	}
}

int main(int argc , char* argv[]){

	clock_t ser_msec;
	int dim1 =  atoi(argv[1]);
	int dim2 =  atoi(argv[2]);
	
	printf("Matrix data type : double\n ");
	printf("dimentions = (%d , %d) ,Tile width = %d",dim1,dim2,TILE_WIDTH);
	
	int size = dim1*dim2*sizeof(Type);

	Type *in_ = (Type*)malloc(size);
	Type *out = (Type*)malloc(size);
	Type *ref = (Type*)malloc(size);
	
	// init matrix
	initial_matirx(in_,dim1,dim2);

	Type *d_in , *d_out;

	hipMalloc(&d_in, size);
	hipMalloc(&d_out, size);
	hipMemcpyAsync(d_in, in_, size, hipMemcpyHostToDevice);
	hipMalloc((Type**)&d_out,size);
	hipMemset(d_out,0,size);

	// init kernel
	int threadNumX = TILE_WIDTH;
	int threadNumY = TILE_WIDTH;
	int blockNumX = dim1 / TILE_WIDTH + (dim1 % TILE_WIDTH == 0 ? 0 : 1 );
	int blockNumY = dim2 / TILE_WIDTH + (dim2 % TILE_WIDTH == 0 ? 0 : 1 );

	dim3 blockSize(threadNumX,threadNumY);
	dim3 gridSize(blockNumX, blockNumY);

	hipEvent_t start, stop,start_shared, stop_shared;
	float elapsedTime, elapsedTime2;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&start_shared);
	hipEventCreate(&stop_shared);

	if(DEBUG && dim1<50 && dim2<50 ){
		printf("\n*********matrix**********\n");
		print_matrix(in_, dim1, dim2);
	}

	ser_msec = clock();
	transpose_CPU(in_, ref, dim1, dim2);
	ser_msec = (double)(clock() - ser_msec) * 1000000 /CLOCKS_PER_SEC;

	if(DEBUG && dim1<50 && dim2<50 ){	
			printf("\n*********serial**********\n");
			print_matrix(ref, dim2, dim1);
	}
	
	hipEventRecord(start, 0);
	transpose_GPU<<<gridSize,blockSize>>>(d_in,d_out,dim1,dim2);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipMemcpy(out,d_out,size,hipMemcpyDeviceToHost);

	hipEventRecord(start_shared, 0);
	transpose_GPU_shared<<<gridSize,blockSize>>>(d_in,d_out,dim1,dim2);
	hipEventRecord(stop_shared, 0);
	hipEventSynchronize(stop_shared);
	hipEventElapsedTime(&elapsedTime2, start_shared, stop_shared);

	hipMemcpy(out,d_out,size,hipMemcpyDeviceToHost);

	if(DEBUG && dim1<50 && dim2<50 ){
		printf("\n*********parallel**********\n");
		print_matrix(out, dim2, dim1);
	}
	
	printf ("\nTime for the serial: \t %ld ms", ser_msec);
	printf ("\nTime for the NAIVE: \t %f ms and speedup: %f", elapsedTime, ser_msec/elapsedTime);
	printf ("\nTime for the shared: \t %f ms and speedup: %f \n\n", elapsedTime2, ser_msec/elapsedTime2);
	

	// cleanup
	hipFree(d_in);
  hipFree(d_out);
	free(in_);
	free(out);
	free(ref);

	return 0;
}